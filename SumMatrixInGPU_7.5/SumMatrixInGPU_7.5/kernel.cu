#include "hip/hip_runtime.h"

#include "hip/hip_runtime.h"
#include ""

#include <cstdio>
#include <random>
#include <string.h>

#include "eTimer.h"

//Matrix size
#define N 6*1024

__global__ void addKernel(double *c, const double *a, const double *b, const double alpha, const double beta)
{
	int x = blockIdx.x*blockDim.x + threadIdx.x;
	int y = blockIdx.y*blockDim.y + threadIdx.y;
	c[y*N + x] = alpha*a[y*N + x] + beta*b[y*N + x];
}

//TODO liberar la memoria de la GPU y la CPU

int main()
{

	double *A, *B, *C;
	//define weights for matrixes
	double alpha = 0.7;
	double beta = 0.6;

	//init random number generator
	std::default_random_engine generador;
	std::normal_distribution<double> distribucion(0.0, 1.0);

	//allocate space for matrixes
	A = (double*)_aligned_malloc(N*N*sizeof(double), 64);
	B = (double*)_aligned_malloc(N*N*sizeof(double), 64);
	C = (double*)_aligned_malloc(N*N*sizeof(double), 64);

	for (int i = 0; i < N; i++)
	{
		for (int j = 0; j < N; j++)
		{
			A[i*N + j] = distribucion(generador);
			B[i*N + j] = distribucion(generador);
		}
	}

	eTimer *Tcpu = new eTimer(); // timer for time to compute on CPU
	eTimer *THtD = new eTimer(); //timer for time to transfer data to GPU
	eTimer *Tkernel = new eTimer(); //timer for GPU to compute
	eTimer *TDtH = new eTimer(); //timer to transfer data from GPU

	Tcpu->start();
	for (int i = 0; i < N; i++)
	{
		for (int j = 0; j < N; j++)
		{
			C[i*N + j] = alpha*A[i*N + j] + beta * B[i*N + j];
		}
	}
	Tcpu->stop();
	Tcpu->report("CPU");

	for (int i = 0; i < 5; i++) printf("%lf ", C[i]);
	printf("\n\n");

	memset(C, 0, N*N*sizeof(double));
	for (int i = 0; i < 5; i++)
	{
		printf("%lf ", C[i]);
	}
	printf("\n\n");

	/*---------------------------GPU-------------------------------------------*/
	hipError_t cudaStatus;

	//It's supposed to be one and only one GPU. And that's the chosen one.
	cudaStatus = hipSetDevice(0);

	//pointers to GPU memory
	double *dev_A, *dev_B, *dev_C;

	cudaStatus = hipMalloc((void**)&dev_C, N*N*sizeof(double));
	cudaStatus = hipMalloc((void**)&dev_B, N*N*sizeof(double));
	cudaStatus = hipMalloc((void**)&dev_A, N*N*sizeof(double));

	THtD->start();
	cudaStatus = hipMemcpy(dev_A, A, N*N*sizeof(double),hipMemcpyHostToDevice);
	cudaStatus = hipMemcpy(dev_B, B, N*N*sizeof(double), hipMemcpyHostToDevice);
	THtD->stop();
	THtD->report("HostToDevice");

	double AnchoBanda = 2 * N*N*sizeof(double) / THtD->get();
	printf("\nAncho de banda(promedio): %lf GBs\n", AnchoBanda*1.0e-9);
	Tkernel->start();
	dim3 Grid, Block;
	Block.x = 32;
	Block.y = 16;
	Grid.x = N / Block.x;
	Grid.y = N / Block.y;

	addKernel <<< Grid, Block >>>(dev_C, dev_A, dev_B,alpha, beta);

	cudaStatus = hipGetLastError();
	if (cudaStatus != hipSuccess){
		fprintf(stderr, "addKernel lauch failed %s\n", hipGetErrorString(cudaStatus));
		exit(1);
	}
	cudaStatus = hipDeviceSynchronize();
	Tkernel->stop();
	Tkernel->report("Kernel");

	TDtH->start();
	cudaStatus = hipMemcpy(C, dev_C, N*N*sizeof(double), hipMemcpyDeviceToHost);
	TDtH->stop();
	TDtH->report("DeviceToHost");

	for (int i = 0; i < 5; i++)
	{
		printf("%lf ", C[i]);
	}
	printf("\n\n");

	cudaStatus = hipDeviceReset();

	delete Tcpu;
	delete THtD;
	delete Tkernel;
	delete TDtH;

	std::getchar();
	return 0;
}
